#include "hip/hip_runtime.h"
#ifndef BUILD_PYTHON_EXTENSION
#include <torch/extension.h>
#else
#include <torch/torch.h>
#endif

// namespace py = pybind11;
__global__ void example_cuda_kernel(float *x, int size) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        x[index] *= 2;
    }
}

void example_cuda_function(torch::Tensor x) {
    const int threads = 1024;
    const int blocks = (x.numel() + threads - 1) / threads;
    example_cuda_kernel<<<blocks, threads>>>(x.data_ptr<float>(), x.numel());
}

#ifndef BUILD_PYTHON_EXTENSION
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("example_cuda_function", &example_cuda_function, "A simple CUDA example function");
}
#endif

